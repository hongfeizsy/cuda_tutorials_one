#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include "add_functions.cuh"


#define USE_UNIX 0

#if USE_UNIX
#include <sys/time.h>
#include <time.h>

double get_time() {
	struct timeval tv;
	double t;

	gettimeofday(&tv, (struct timezone *)0);
	t = tv.tv_sec + (double)tv.tv_usec * 1e-6;
	
	return t;
}

#else
#include <Windows.h>

double get_time() {
	LARGE_INTEGER timer;
	static LARGE_INTEGER fre;
	static int init = 0;
	double t;

	if (init != 1) {
		QueryPerformanceFrequency(&fre);
		init = 1;
	}

	QueryPerformanceCounter(&timer);
	t = timer.QuadPart * 1.0 / fre.QuadPart;
	
	return t;
}
#endif


int main() {
	int N = 20000000;
	int nbytes = sizeof(float) * N;

	/* 1D block */
	int block_size = 256;

	/* 2D grid */
	int s = ceil(sqrt((N + block_size - 1.0) / block_size));
	dim3 grid = dim3(s, s);
	
	float *dx = NULL, *hx = NULL;
	float *dy = NULL, *hy = NULL;
	float *dz = NULL, *hz = NULL;

	int itr = 30;
	int i;
	double th, td;

	/* allocate GPU memory */
	hipMalloc((void **)&dx, nbytes);
	hipMalloc((void **)&dy, nbytes);
	hipMalloc((void **)&dz, nbytes);

	if (dx == NULL || dy == NULL || dz == NULL) {
		printf("couldn't allocate GPU memory\n");
		return -1;
	}
	printf("allocated %.2f MB on GPU\n", nbytes / (1024.0 * 1024.0));

	/* allocate CPU memory */
	hx = (float*)malloc(nbytes);
	hy = (float*)malloc(nbytes);
	hz = (float*)malloc(nbytes);

	/* init */
	for (int i = 0; i < N; i++) {
		hx[i] = 1;
		hy[i] = 1;
		hz[i] = 1;
	}

	/* copy data to GPU */
	hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(dy, hy, nbytes, hipMemcpyHostToDevice);
	hipMemcpy(dz, hz, nbytes, hipMemcpyHostToDevice);

	/* warm up GPU */
	warmup();

	/* call GPU */
	hipDeviceSynchronize();
	td = get_time();
	for (int i = 0; i < itr; i++) vec_add<<<grid, block_size >>> (dx, dy, dz, N);
	hipDeviceSynchronize();
	td = get_time() - td;

	/* call CPU */
	th = get_time();
	for (int i = 0; i < itr; i++) vec_add_host(hx, hy, hz, N);
	th = get_time() - th;
	float temp = hz[0];

	printf("GPU time: %e, CPU time: %e, speedup: %g\n", td, th, th / td);
	free(hx);
	free(hy);
	free(hz);
	hipFree(hx);
	hipFree(hy);
	hipFree(hz);

	return 0;
}





//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
